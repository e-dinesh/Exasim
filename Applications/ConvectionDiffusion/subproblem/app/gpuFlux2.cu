
#include <hip/hip_runtime.h>
template <typename T>  __device__  void devicegpuFlux2(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i<ng) {
		T udg2 = udg[1*ng+i];
		T udg3 = udg[2*ng+i];
		f[0*ng+i] = udg2;
		f[1*ng+i] = udg3;
		i += blockDim.x * gridDim.x;
	}
}

template <typename T> __global__ void kernelgpuFlux2(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	devicegpuFlux2(f, xdg, udg, odg, wdg, uinf, param, time, modelnumber, ng, nc, ncu, nd, ncx, nco, ncw);
}

template <typename T> void gpuFlux2(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	int blockDim = 256;
	int gridDim = (ng + blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	kernelgpuFlux2<<<gridDim, blockDim>>>(f, xdg, udg, odg, wdg, uinf, param, time, modelnumber, ng, nc, ncu, nd, ncx, nco, ncw);
}

template void gpuFlux2(double *, double *, double *, double *, double *, double *, double *, double, int, int, int, int, int, int, int, int);
template void gpuFlux2(float *, float *, float *, float *, float *, float *, float *, float, int, int, int, int, int, int, int, int);
